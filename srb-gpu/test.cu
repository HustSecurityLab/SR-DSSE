#include "hip/hip_runtime.h"
#include </usr/local/include/cuFHE/cufhe_gpu.cuh>
#include <iostream>
#include <time.h>
#include <vector>
#include <cassert>
#include <string>
#include <string.h>
#include <stdint.h>
#include "openssl/md5.h"  
#include "openssl/sha.h" 
#include "openssl/hmac.h"
#include "openssl/evp.h"
#include <iomanip>
#include <map>
#include <math.h>
#include <sstream>
#include <gmp.h>
#include <gmpxx.h>
#include <sys/time.h>
#include <fstream>
#include <stdio.h>
#include <inttypes.h>
#include <string.h>
#include <bitset>
#include <stack> 
#include <unordered_map>

#define MAX_FILE_LEN 1000
#define LEN_OF_SEED 10
#define MAX_TIMES 20

using namespace std;
using namespace cufhe;

float totalupdatetime = 0;
float clientsearch = 0;
float serversearch = 0;
float totalsearch = 0;

string num2str(int a)
{
	string str;
    stringstream ss;
    ss << a;
    ss >> str;
    return str;
}

//md5
void md5(const string &srcStr, string &encodedHexStr)
{
	unsigned char mdStr[33] = { 0 };
	MD5((const unsigned char *)srcStr.c_str(), srcStr.length(), mdStr);// 调用md5哈希
	string encodedStr = std::string((const char *)mdStr);// 哈希后的字符串
	char buf[65] = { 0 };
	char tmp[3] = { 0 };
	for (int i = 0; i < 32; i++)// 哈希后的十六进制串 32字节  
	{
		sprintf(tmp, "%02x", mdStr[i]);
		strcat(buf, tmp);
	}
	buf[32] = '\0'; // 后面都是0，从32字节截断  
	encodedHexStr = std::string(buf);
}

string sha256(const string str)
{
	char buf[10000];
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, str.c_str(), str.size());
    SHA256_Final(hash, &sha256);
    std::string NewString = "";
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(buf,"%02x",hash[i]);
        NewString = NewString + buf;
    }
	return NewString;
}

string sha512(const string str)
{
	char buf[10000];
    unsigned char hash[SHA512_DIGEST_LENGTH];
    SHA512_CTX sha512;
    SHA512_Init(&sha512);
    SHA512_Update(&sha512, str.c_str(), str.size());
    SHA512_Final(hash, &sha512);
    std::string NewString = "";
    for(int i = 0; i < SHA512_DIGEST_LENGTH; i++)
    {
        sprintf(buf,"%02x",hash[i]);
        NewString = NewString + buf;
    }
	return NewString;
}

string newxor(string s1, string s2)
{
	mpz_t ss1,ss2,ssf;
	char* result = new char[1000];
	mpz_init(ss1);
	mpz_init(ss2);
	mpz_init(ssf);
	mpz_set_str(ss1, s1.c_str(), 16);
	mpz_set_str(ss2, s2.c_str(), 16);
	mpz_xor(ssf,ss1,ss2);
	mpz_get_str(result, 16, ssf);
	string fr = result;

	for(int i = fr.length();i<128;i++)
	{
		fr = "0"+fr;
	}

	return fr;
}

string rand_str(const int len)  /*参数为字符串的长度*/
{
    /*初始化*/
    string str;                 /*声明用来保存随机字符串的str*/
    char c;                     /*声明字符c，用来保存随机生成的字符*/
    int idx;                    /*用来循环的变量*/
	srand((int)time(0));
    for(idx = 0;idx < len;idx ++)
    {
        c = 'a' + rand()%26;
        str.push_back(c);       /*push_back()是string类尾插函数。这里插入随机字符c*/
    }
    return str;                 /*返回生成的随机字符串*/
}

string rand_str2(const int len)
{
	clock_t time = clock();
    gmp_randstate_t grt;
    gmp_randinit_default(grt);
    gmp_randseed_ui(grt, time);

	mpz_t num;
	mpz_init2(num,len);
	mpz_urandomb(num,grt,len);

	char* numstr = new char[len];
	mpz_get_str(numstr,10,num);

	string sp = numstr;
	return sp;
}

string hmac256(string key, string msg)
{
	stringstream ss;
    HMAC_CTX *ctx;
	ctx = HMAC_CTX_new();
    unsigned int  len;
    unsigned char out[EVP_MAX_MD_SIZE];
    HMAC_Init_ex(ctx, key.c_str(), key.length(), EVP_sha256(),NULL);
    HMAC_Update(ctx, (unsigned char*)msg.c_str(), msg.length());
    HMAC_Final(ctx, out, &len);
	HMAC_CTX_free(ctx);
    for (unsigned int i = 0;  i < len;  i++)
    {
        ss << setw(2) << setfill('0') << hex << static_cast<int> (out[i]);
    }
    return ss.str();
}

string hmac512(string key, string msg)
{
	stringstream ss;
    HMAC_CTX *ctx;
	ctx = HMAC_CTX_new();
    unsigned int  len;
    unsigned char out[EVP_MAX_MD_SIZE];
    HMAC_Init_ex(ctx, key.c_str(), key.length(), EVP_sha512(),NULL);
    HMAC_Update(ctx, (unsigned char*)msg.c_str(), msg.length());
    HMAC_Final(ctx, out, &len);
	HMAC_CTX_free(ctx);
    for (unsigned int i = 0;  i < len;  i++)
    {
        ss << setw(2) << setfill('0') << hex << static_cast<int> (out[i]);
    }
    return ss.str();
}

string H_1(string key, string msg)
{
	return hmac256(key, msg);
}

string H_2(string key, string msg)
{
	return hmac512(key, msg);
}

string PRF(string key, string msg)
{
	stringstream ss;
    HMAC_CTX *ctx;
	ctx = HMAC_CTX_new();
    unsigned int  len;
    unsigned char out[EVP_MAX_MD_SIZE];
    HMAC_Init_ex(ctx, key.c_str(), key.length(), EVP_md5(),NULL);
    HMAC_Update(ctx, (unsigned char*)msg.c_str(), msg.length());
    HMAC_Final(ctx, out, &len);
	HMAC_CTX_free(ctx);
    for (unsigned int i = 0;  i < len;  i++)
    {
        ss << setw(2) << setfill('0') << hex << static_cast<int> (out[i]);
    }
    return ss.str();
}



class Client
{
	public:
	string ST_c;
    int counter;
	string keyword;

	string K;

	PriKey pri_key; // private key
	PubKey pub_key; // public key

	int init(string word)
    {
    	counter = -1;
        string in = rand_str(10);
		string in2 = rand_str(10);

        ST_c = sha512(in2);
		K = sha256(in);

		keyword = word;

		SetSeed(); // set random seed
		KeyGen(pub_key, pri_key);
		Initialize(pub_key);
        return 1;
    }
};

class CTelement
{
	public:
	string C_ST_C;
	Ctxt* Vc = new Ctxt[MAX_FILE_LEN];
	Ctxt* Vd = new Ctxt[MAX_FILE_LEN];
};

void Update(Client &client,unordered_map<string,CTelement> &Server)
{
	string in = rand_str2(LEN_OF_SEED);
	
	clock_t time = clock();
    gmp_randstate_t grt;
    gmp_randinit_default(grt);
    gmp_randseed_ui(grt, time);

    mpz_t va,vb;

    mpz_init2(va, MAX_FILE_LEN);               //random operation a and operation b
    mpz_init2(vb, MAX_FILE_LEN);
    mpz_urandomb(va,grt,MAX_FILE_LEN);
    mpz_urandomb(vb,grt,MAX_FILE_LEN);

	Ptxt* pt = new Ptxt[MAX_FILE_LEN];
	Ptxt* pt2 = new Ptxt[MAX_FILE_LEN];
	for(int i = 0;i<MAX_FILE_LEN;i++)
    {
        pt[i] = mpz_tstbit(va, i);
        pt2[i] = mpz_tstbit(vb, i);
    }


	float et;
  	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	hipEventRecord(start, 0);

	//Generate token
	string ST_new = "";
    ST_new = sha512(in);

	string Keys = PRF(client.K, client.keyword);

	client.counter++;
	string UT_new = H_1(Keys, ST_new);
	string C_ST_c = newxor(H_2(Keys, ST_new),client.ST_c);
	client.ST_c = ST_new;

	CTelement cc;
	cc.C_ST_C = C_ST_c;

	for(int i = 0;i<MAX_FILE_LEN;i++)
    {
        Encrypt(cc.Vc[i], pt[i], client.pri_key);
		Encrypt(cc.Vd[i], pt2[i], client.pri_key);
    }

	Server[UT_new] = cc;

	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&et, start, stop);

	totalupdatetime+=et;

	return;
}

void Search(Client &client, unordered_map<string,CTelement> &Server)
{
	int timecount = client.counter;
	float et1, et2, et3;
	hipEvent_t start1, stop1, start2, stop2, start3, stop3;
  	hipEventCreate(&start1);
  	hipEventCreate(&stop1);
   
	string in = rand_str2(LEN_OF_SEED);
	string in2 = rand_str2(LEN_OF_SEED+1);

	Ptxt* pt = new Ptxt[MAX_FILE_LEN];
	Ctxt* Vc = new Ctxt[MAX_FILE_LEN];
	for(int i = 0;i<MAX_FILE_LEN;i++)
    {
        pt[i] = 0;
    }

	hipEventRecord(start1, 0);

	//string new_ST = sha256(in2);

	string Keys = PRF(client.K, client.keyword);

	string ST_temp = client.ST_c;
	string UT_c = "";
	//string New_UT = H_1(Keys, new_ST);
	//string new_CT = "0000000000000000";  //new token generation

	//client.counter++;

	// CTelement cc;
	// cc.C_ST_C = new_CT;

	for(int i = 0;i<MAX_FILE_LEN;i++)
    {
        Encrypt(Vc[i], pt[i], client.pri_key);
    }

	hipEventRecord(stop1, 0);
  	hipEventSynchronize(stop1);
  	hipEventElapsedTime(&et1, start1, stop1);

	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	uint32_t kNumSMs = prop.multiProcessorCount;
	cout<<"Number of process: "<<kNumSMs<<endl;

	hipEventCreate(&start2);
  	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);

	stack<Ctxt*> Vaa;
	stack<Ctxt*> Vbb;


	Stream* st = new Stream[kNumSMs];
	for (int i = 0; i < kNumSMs; i ++)
	{
		st[i].Create();
	}

	Ctxt* tmp1 = new Ctxt[MAX_FILE_LEN];
	Ctxt* tmp2 = new Ctxt[MAX_FILE_LEN];

	for(int q = timecount;q>=0;q--)
	{
		string UT = H_1(Keys, ST_temp);
		if(q==timecount)
		{
			UT_c = UT;
		}
		string C_ST = Server[UT].C_ST_C;
		// cout<<"The Search UT is: "<<UT<<endl;
		// cout<<"The Search shield code is: "<<C_ST<<endl;
		ST_temp = newxor(C_ST, H_2(Keys, ST_temp));

		Vaa.push(Server[UT].Vc);
		Vbb.push(Server[UT].Vd);

		if(C_ST=="NULL")
		{
			break;
		}
	}

	while(!Vaa.empty())
	{
		Synchronize();
		for(int i = 0; i<MAX_FILE_LEN;i++)
        {
			And(tmp1[i], Vaa.top()[i], Vbb.top()[i],st[i % kNumSMs]);
		}
		for(int i = 0; i<MAX_FILE_LEN;i++)
        {
			Not(tmp2[i], Vaa.top()[i], st[i % kNumSMs]);
		}
		for(int i = 0; i<MAX_FILE_LEN;i++)
        {
			And(Vc[i], Vc[i], tmp2[i], st[i % kNumSMs]);
		}
		for(int i = 0; i<MAX_FILE_LEN;i++)
        {
			Xor(Vc[i], Vc[i], tmp1[i], st[i % kNumSMs]);
		}
		Synchronize();
		Vaa.pop();
		Vbb.pop();
	}

	Synchronize();
	Server[UT_c].C_ST_C = "NULL";
	for(int i = 0; i<MAX_FILE_LEN;i++)
    {
		Copy(Server[UT_c].Vc[i], Vc[i], st[i % kNumSMs]);
		Copy(Server[UT_c].Vd[i], Vc[i], st[i % kNumSMs]);
	}
	Synchronize();

	hipEventRecord(stop2, 0);
  	hipEventSynchronize(stop2);
  	hipEventElapsedTime(&et2, start2, stop2);

	
	hipEventCreate(&start3);
  	hipEventCreate(&stop3);
	hipEventRecord(start3, 0);
	Ptxt* re1 = new Ptxt[MAX_FILE_LEN];
	for(int p = MAX_FILE_LEN-1;p>=0;p--)
	{
		Decrypt(re1[p], Vc[p], client.pri_key);
		//cout<<re1[p].message_;
	}

	hipEventRecord(stop3, 0);
  	hipEventSynchronize(stop3);
  	hipEventElapsedTime(&et3, start3, stop3);


	clientsearch = et1+et3;
	serversearch = et2;
	totalsearch = clientsearch+serversearch;
	
	return;
}

 
int main() 
{
	map<string, Client> CT;
	string keyword = "dhc";    //get from erone data
    Client c;
    c.init(keyword);
	//CT[word] = c;

    unordered_map<string,CTelement> Server;
    for(int times = 0;times<MAX_TIMES;times++)   
	{
        Update(c,Server);
	}
	cout<<"The total update time is: "<<totalupdatetime<<"ms"<<endl;
	cout<<"The average update time is: "<<totalupdatetime/MAX_TIMES<<"ms"<<endl;
	Search(c,Server);
	cout<<"The client search time is: "<<clientsearch<<"ms"<<endl;
	cout<<"The server search time is: "<<serversearch/1000<<"s"<<endl;
	cout<<"The total search time is: "<<totalsearch/1000<<"s"<<endl;
	return 0;
}
